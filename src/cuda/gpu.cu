#include "hip/hip_runtime.h"
/*  This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>. */
extern "C" {
#include <ROOT-Sim.h>
#include <ftl/ftl.h>
#include <arch/timer.h>

typedef unsigned rid_t;
extern __thread rid_t rid;
extern timer_uint gvt_timer;
extern struct simulation_configuration global_config;
}

#include "gpu.h"
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "kernels.h"
#include "nelder_mead_3d.h"
#include "statistics.cu"

/* Global variables*/
__device__ uint g_n_nodes;
__device__ uint g_n_lps;
__device__ uint g_nodes_per_lp;

static uint nodes_per_lp;
static uint n_nodes;
static uint n_lps;

uint events_per_node;
static uint states_per_node;
static uint antimsgs_per_node;

static float inactive_lps_percent;
static int window_size;

uint threads_per_block;
uint n_blocks;
timer_uint gpu_gvt_timer;


extern "C" uint get_n_nodes(){return n_nodes;}
extern "C" uint get_n_lps(){return n_lps;}
extern "C" uint get_n_nodes_per_lp(){return nodes_per_lp;}
extern "C" uint get_n_blocks(){return n_blocks;}


/* Private functions */
// XXX Should be either static or moved to an internal header
char *get_time();
size_t get_free_memory();
void print_size(size_t size);
uint get_number_blocks(uint n_threads);
int get_gvt(int *d_ts_temp);
char change_nodes_per_lp(uint target_value_log, char *d_can_split);
void merge_lps();
char split_lps(char *d_can_split);

static float initial_p1[] = {0.6, 1, 0.6, 0.6};

/* For PHOLD model with lambda = 1 or 100 */
//	float	initial_p2[] = {1000, 1000, 5000, 1000};
/* For PHOLD model with lambda = 10000 */
static float initial_p2[] = {1000, 1000, 50000, 1000};
/* For Kademlia models */
//	float	initial_p2[] = { 100,  100, 1000,  100};

static float initial_p3[] = {0.5, 0.5, 0.5, 4.5};

extern "C"
bool gpu_is_available(void)
{
	int devices = 0;
	hipError_t ret = hipGetDeviceCount(&devices);

	if (devices > 0 && ret == hipSuccess)
		return true;
	printf("cannod find any CUDA device\n");
	exit(1);
	return false;
}


static void magic_numbers(int n)
{
	nodes_per_lp = 8;//pow(2, (int)initial_p3[0]);
	n_nodes = n;
	n_lps = n_nodes / nodes_per_lp;

	/* For PHOLD models */
	events_per_node = 50;
	states_per_node = 30;
	antimsgs_per_node = 30;

	/* For Kademlia models */
	//	events_per_node = 30;
	//	states_per_node = 15;
	//	antimsgs_per_node = 30;

	inactive_lps_percent = initial_p1[0];
	window_size = initial_p2[0];


#if(OPTM_SYNC == 0)
	states_per_node = 0;
	antimsgs_per_node = 0;
#endif

#if(ALLOW_ME == 0)
	inactive_lps_percent = 0;
#endif

	threads_per_block = 256;
	n_blocks = get_number_blocks(n_lps);
}

/* For PHOLD models
 * Parameters are: population, lookahead, mean
 */
static int model_params[] = {-1, 1000, 10000}; // First parameter is set in configure_gpu()
static uint n_params = 3;

// XXX All this should me moved to a configuration struct

static int *d_model_params;
static int *d_lookahead;
static int *d_ts_temp;
static uint *d_n_events_cmt;
static uint *d_inac_1, *d_inac_2, *d_inac_3;
static uint *d_inac_4, *d_inac_5, *d_inac_6;
static char *d_rollback_performed;
static char *d_can_split;

static int h_lookahead;
static uint h_n_events_cmt;
static uint h_inac_1, h_inac_2, h_inac_3;
static uint h_inac_4, h_inac_5, h_inac_6;
static char h_rollback_performed;

// Measurement
static char warm_up = 0;
static int setup = 0;
static char flag_continue = 0;
static float total_events = 0;
static float n_events_since_change = 0;
static float max_rate = 0;
static float min_rate = 1000;

static hipEvent_t start, stop, start_1, stop_1, start_2, stop_2;
static float exec_time;


#include <locale.h>
extern "C"
bool gpu_configure(lp_id_t n_lps)
{
	// int n = 1048576; // XXX Number of nodes in the original code?! Is this LPs?
	int n = n_lps;
	model_params[0] = n_lps;

	magic_numbers(n);

	hipDeviceReset();

	// hipMalloc
	if(hipMalloc(&d_model_params, sizeof(int) * n_params) != hipSuccess ||
	    hipMalloc(&d_lookahead, sizeof(int)) != hipSuccess ||
	    hipMalloc(&d_ts_temp, sizeof(int) * n_blocks) != hipSuccess ||
	    hipMalloc(&d_n_events_cmt, sizeof(uint)) != hipSuccess ||
	    hipMalloc(&d_inac_1, sizeof(uint)) != hipSuccess ||
	    hipMalloc(&d_inac_2, sizeof(uint)) != hipSuccess ||
	    hipMalloc(&d_inac_3, sizeof(uint)) != hipSuccess ||
	    hipMalloc(&d_inac_4, sizeof(uint)) != hipSuccess ||
	    hipMalloc(&d_inac_5, sizeof(uint)) != hipSuccess ||
	    hipMalloc(&d_inac_6, sizeof(uint)) != hipSuccess ||
	    hipMalloc(&d_can_split, sizeof(char)) != hipSuccess ||
	    hipMalloc(&d_rollback_performed, sizeof(char)) != hipSuccess) {
		return false;
	}

	hipMemcpy(d_model_params, model_params, sizeof(int) * n_params, hipMemcpyHostToDevice);

	h_n_events_cmt = 0;
	hipMemcpy(d_n_events_cmt, &h_n_events_cmt, sizeof(uint), hipMemcpyHostToDevice);

	h_rollback_performed = 0;
	hipMemcpy(d_rollback_performed, &h_rollback_performed, sizeof(char), hipMemcpyHostToDevice);

	char res = 0;
	res += malloc_nodes(n_nodes);
	res += malloc_queues(n_nodes, events_per_node, states_per_node, antimsgs_per_node);

	if(res != 2) {
		free_nodes();
		free_queues();

		printf("ERROR: Memory not enough.\n");
		return true;
	}

	// Initialization
	kernel_set_params<<<1, 1>>>(n_nodes, n_lps, nodes_per_lp, events_per_node, states_per_node,
	    antimsgs_per_node, d_model_params, n_params);

	kernel_get_lookahead<<<1, 1>>>(d_lookahead);
	hipMemcpy(&h_lookahead, d_lookahead, sizeof(int), hipMemcpyDeviceToHost);

	kernel_init_queues<<<n_blocks, threads_per_block>>>();
	kernel_init_nodes<<<n_blocks, threads_per_block>>>();
	kernel_sort_event_queues<<<n_blocks, threads_per_block>>>();

	// Time measurement
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start_1);
	hipEventCreate(&stop_1);
	hipEventCreate(&start_2);
	hipEventCreate(&stop_2);

	// Simulation
	hipDeviceSynchronize();

	hipEventRecord(start);
	hipEventRecord(start_2);
			

	return true;
}

extern "C"
thrd_ret_t THREAD_CALL_CONV gpu_main_loop(void *args)
{
	(void)args;
	rid = (uintptr_t)args;
	gpu_gvt_timer = timer_new();
//    setlocale(LC_NUMERIC, "");
	follow_the_leader(0);	
	int gvt = 0, prev_gvt = 0;

	while(!sim_can_end()) {
		// Get minimal timestamp of all next events
		gvt = get_gvt(d_ts_temp);

		timer_uint t = timer_new();
		if(global_config.gvt_period < t - gpu_gvt_timer){
			printf("\t\t\t\t\tGPU GVT  %.2g, (%.2g, %u), %.2g, %lu, %lf, %f\n", (float)gvt, (float)(gvt - prev_gvt), global_config.gvt_period,
			(float)(gvt - prev_gvt) / global_config.gvt_period, t/1000, (double)gvt, gimme_current_time_please());
            prev_gvt = gvt;
			fflush(stdout);
			gpu_gvt_timer = t;

			follow_the_leader((simtime_t)gvt*1.0);

		}
		
        	
		// Delete past events
		h_n_events_cmt = 0;
		hipMemcpy(d_n_events_cmt, &h_n_events_cmt, sizeof(uint), hipMemcpyHostToDevice);

		kernel_clean_queues<<<n_blocks, threads_per_block>>>(gvt, d_n_events_cmt);

		hipMemcpy(&h_n_events_cmt, d_n_events_cmt, sizeof(uint), hipMemcpyDeviceToHost);
		total_events += h_n_events_cmt;
		n_events_since_change += h_n_events_cmt;

		if( ((float)gvt) > global_config.termination_time) {
			gpu_ended();
		}
		
		
		// Change parameters
		//hipEventRecord(stop_2);
		//hipEventSynchronize(stop_2);
		//hipEventElapsedTime(&exec_time, start_2, stop_2);
		//float current_rate = n_events_since_change / exec_time / 1000;

		//if(warm_up == 0 && exec_time > 200) {
		//	warm_up = 1;

		//	n_events_since_change = 0;
		//	hipEventRecord(start_2);

		//	flag_continue = 1;
		//} else if(warm_up == 1 && exec_time > 300) {
		//	warm_up = 0;

		//	//			printf("%4.2f %6d %2u %10.3f %10.0f %10d",
		//	//				inactive_lps_percent, window_size,
		//	//				nodes_per_lp, current_rate, total_events, gvt);

		//	if(current_rate < min_rate) {
		//		min_rate = current_rate;
		//		//				printf(" --> MIN");
		//	} else if(current_rate > max_rate) {
		//		max_rate = current_rate;
		//		//				printf(" --> MAX");
		//	}

		//	//			printf("\n");

		//	if(setup < 4) {
		//		nm_start(setup, initial_p1[setup], initial_p2[setup], initial_p3[setup], current_rate);

		//		setup++;
		//		if(setup < 4) {
		//			inactive_lps_percent = initial_p1[setup];
		//			window_size = initial_p2[setup];
		//			change_nodes_per_lp(initial_p3[setup], d_can_split);

		//			n_events_since_change = 0;
		//			hipEventRecord(start_2);
		//			flag_continue = 1;
		//		}
		//	}

		//	if(flag_continue == 0) {
		//		float next_p1, next_p2, next_p3;

		//		char flag = 0;

		//		while(flag == 0) {
		//			nm_get_next_point(current_rate, &next_p1, &next_p2, &next_p3);

		//			flag = 1;

		//			if(next_p1 < 0 || next_p1 > 1 || next_p2 < 0 || next_p3 < 0) {
		//				flag = 0;
		//				current_rate = 0;
		//				continue;
		//			}

		//			inactive_lps_percent = next_p1;
		//			window_size = next_p2;
		//			char res = change_nodes_per_lp(next_p3, d_can_split);
		//			if(res == 0) {
		//				flag = 0;
		//				current_rate = 0;
		//				continue;
		//			}
		//		}

		//		n_events_since_change = 0;
		//		hipEventRecord(start_2);

		//		flag_continue = 1;
		//	}
		//}

		//if(flag_continue == 1) {
		//	flag_continue = 0;
		//	continue;
		//}

		// Handle next event
		while(1) {
			h_inac_1 = h_inac_2 = h_inac_3 = 0;
			h_inac_4 = h_inac_5 = h_inac_6 = 0;
			hipMemcpy(d_inac_1, &h_inac_1, sizeof(uint), hipMemcpyHostToDevice);
			hipMemcpy(d_inac_2, &h_inac_2, sizeof(uint), hipMemcpyHostToDevice);
			hipMemcpy(d_inac_3, &h_inac_3, sizeof(uint), hipMemcpyHostToDevice);
			hipMemcpy(d_inac_4, &h_inac_4, sizeof(uint), hipMemcpyHostToDevice);
			hipMemcpy(d_inac_5, &h_inac_5, sizeof(uint), hipMemcpyHostToDevice);
			hipMemcpy(d_inac_6, &h_inac_6, sizeof(uint), hipMemcpyHostToDevice);

#if(OPTM_SYNC == 1)
			kernel_handle_next_event<<<n_blocks, threads_per_block>>>(gvt, window_size, d_inac_1, d_inac_2,
			    d_inac_3, d_inac_4, d_inac_5, d_inac_6);
#else
			kernel_handle_next_event<<<n_blocks, threads_per_block>>>(gvt, h_lookahead, d_inac_1, d_inac_2,
			    d_inac_3, d_inac_4, d_inac_5, d_inac_6);
#endif

			hipMemcpy(&h_inac_1, d_inac_1, sizeof(uint), hipMemcpyDeviceToHost);
			hipMemcpy(&h_inac_2, d_inac_2, sizeof(uint), hipMemcpyDeviceToHost);
			hipMemcpy(&h_inac_3, d_inac_3, sizeof(uint), hipMemcpyDeviceToHost);
			hipMemcpy(&h_inac_4, d_inac_4, sizeof(uint), hipMemcpyDeviceToHost);
			hipMemcpy(&h_inac_5, d_inac_5, sizeof(uint), hipMemcpyDeviceToHost);
			hipMemcpy(&h_inac_6, d_inac_6, sizeof(uint), hipMemcpyDeviceToHost);

#if(ALLOW_ME == 1)
			uint inac = h_inac_1 + h_inac_2 + h_inac_3 + h_inac_4 + h_inac_5 + h_inac_6;
			if(inac >= n_lps * inactive_lps_percent) {
				break;
			}
#else
			break;
#endif
		}

#if(OPTM_SYNC == 1)
		// Roll back
		while(1) {
			h_rollback_performed = 0;
			hipMemcpy(d_rollback_performed, &h_rollback_performed, sizeof(char), hipMemcpyHostToDevice);

			kernel_roll_back<<<n_blocks, threads_per_block>>>(d_rollback_performed);

			hipMemcpy(&h_rollback_performed, d_rollback_performed, sizeof(char), hipMemcpyDeviceToHost);
			if(h_rollback_performed == 0) {
				break;
			}
		}
#endif

		hipError_t err = hipGetLastError();
		if(err != hipSuccess) {
			printf("FATAL ERROR: %s\n", hipGetErrorString(err));
			exit(1);
		}

		// Sort
		kernel_sort_event_queues<<<n_blocks, threads_per_block>>>();
	}
	return NULL;
}

extern "C"
void gpu_stop(void)
{
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&exec_time, start, stop);

	float avg_rate = total_events / exec_time / 1000;

	free_queues();
	free_nodes();

	hipFree(d_model_params);
	hipFree(d_lookahead);
	hipFree(d_ts_temp);
	hipFree(d_n_events_cmt);
	hipFree(d_inac_1);
	hipFree(d_inac_2);
	hipFree(d_inac_3);
	hipFree(d_inac_4);
	hipFree(d_inac_5);
	hipFree(d_inac_6);
	hipFree(d_rollback_performed);

	printf("%5.2f %5.2f %5.2f\n",min_rate, avg_rate, max_rate);
}

char *get_time()
{
	time_t now = time(NULL);
	char *time_str = ctime(&now);
	time_str[strlen(time_str) - 1] = 0;
	return time_str;
}

size_t get_free_memory()
{
	size_t free, total;
	hipMemGetInfo(&free, &total);
	return free;
}

void print_size(size_t size)
{
	size_t numbers[3];
	numbers[0] = size / 1000000;
	numbers[1] = (size % 1000000) / 1000;
	numbers[2] = size % 1000;

	printf("%zu,%03zu,%03zu", numbers[0], numbers[1], numbers[2]);
}

uint get_number_blocks(uint n_threads)
{
	return n_threads / threads_per_block + (n_threads % threads_per_block == 0 ? 0 : 1);
}

int get_gvt(int *d_ts_temp)
{
	kernel_get_gvt_1<<<n_blocks, threads_per_block, threads_per_block * sizeof(int)>>>(d_ts_temp);

	uint next_n_blocks = get_number_blocks(n_blocks);
	uint n_left = n_blocks;
	uint distance = 1;

	while(n_left != 1) {
		kernel_get_gvt_2<<<next_n_blocks, threads_per_block, threads_per_block * sizeof(int)>>>(d_ts_temp,
		    n_left, distance);
		n_left = next_n_blocks;
		next_n_blocks = get_number_blocks(next_n_blocks);
		distance *= threads_per_block;
	}

	int gvt;
	hipMemcpy(&gvt, d_ts_temp, sizeof(int), hipMemcpyDeviceToHost);

	return gvt;
}

char change_nodes_per_lp(uint target_value_log, char *d_can_split)
{
	uint target = pow(2, target_value_log);
	if(nodes_per_lp == target) {
		return 1;
	}

	uint nodes_per_lp_before = nodes_per_lp;

	if(nodes_per_lp > target) {
		while(nodes_per_lp != target) {
			if(split_lps(d_can_split) == 0) {
				while(nodes_per_lp != nodes_per_lp_before) {
					merge_lps();
				}

				return 0;
			}
		}
	} else {
		while(nodes_per_lp != target) {
			merge_lps();
		}
	}

	return 1;
}

void merge_lps()
{
#if(OPTM_SYNC == 1)
	kernel_roll_back_all<<<n_blocks, threads_per_block>>>();
	kernel_sort_event_queues<<<n_blocks, threads_per_block>>>();
#endif

	kernel_rotate_queues<<<n_blocks, threads_per_block>>>();
	kernel_merge_queues<<<n_blocks, threads_per_block>>>();
	kernel_adjust_queues_after_merge<<<n_blocks, threads_per_block>>>();
	kernel_set_params_after_merge<<<1, 1>>>();

	nodes_per_lp = nodes_per_lp * 2;
	n_lps = n_lps / 2 + (n_lps % 2 == 0 ? 0 : 1);
	n_blocks = get_number_blocks(n_lps);

	hipDeviceSynchronize();
	kernel_sort_event_queues<<<n_blocks, threads_per_block>>>();
}

char split_lps(char *d_can_split)
{
	char h_can_split = 1;
	hipMemcpy(d_can_split, &h_can_split, sizeof(char), hipMemcpyHostToDevice);
	kernel_check_queues_before_split<<<n_blocks, threads_per_block>>>(d_can_split);
	hipMemcpy(&h_can_split, d_can_split, sizeof(char), hipMemcpyDeviceToHost);

	if(h_can_split == 0) {
		//		printf("IMPOSSBILE TO SPLIT.\n");
		return 0;
	}

#if(OPTM_SYNC == 1)
	kernel_roll_back_all<<<n_blocks, threads_per_block>>>();
	kernel_sort_event_queues<<<n_blocks, threads_per_block>>>();
#endif

	kernel_rotate_queues<<<n_blocks, threads_per_block>>>();
	kernel_adjust_queues_before_split<<<n_blocks, threads_per_block>>>();
	kernel_split_queues<<<n_blocks, threads_per_block>>>();
	kernel_set_params_after_split<<<1, 1>>>();

	nodes_per_lp = nodes_per_lp / 2;
	n_lps = n_lps * 2;
	n_blocks = get_number_blocks(n_lps);

	hipDeviceSynchronize();
	kernel_sort_event_queues<<<n_blocks, threads_per_block>>>();

	return 1;
}
