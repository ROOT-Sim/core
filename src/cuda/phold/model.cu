/*  This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>. */


#include <cuda/cuda_gpu.h>
#include <cuda/queues.h>
#include <cuda/kernels.h>
#include "model.h"
#include "settings.h"

__device__ static Nodes nodes;

__device__ static uint	population;
__device__ static int	lookahead;
__device__ static int	mean;


hiprandState_t *simulation_snapshot;

char malloc_nodes(uint n_nodes) {
	hipError_t err;

	Nodes h_nodes;
	simulation_snapshot = (hiprandState_t*) malloc(sizeof(hiprandState_t)*n_nodes);
	if(!simulation_snapshot) {printf("no memory for HOST side model state\n"); exit(1); }
	
	err = hipMalloc(&(h_nodes.cr_state), sizeof(hiprandState_t) * n_nodes);
	if (err != hipSuccess) { return 0; }
	hipMemcpyToSymbol(HIP_SYMBOL(nodes), &h_nodes, sizeof(Nodes));

	return 1;
}

void copy_nodes_from_host(uint n_nodes) {
	Nodes h_nodes;
	hipMemcpyFromSymbol(&h_nodes, HIP_SYMBOL(nodes), sizeof(Nodes));
	hipMemcpy(h_nodes.cr_state, simulation_snapshot, sizeof(hiprandState_t) * n_nodes, hipMemcpyHostToDevice);
}


void copy_nodes_to_host(uint n_nodes) {
	Nodes h_nodes;
	hipMemcpyFromSymbol(&h_nodes, HIP_SYMBOL(nodes), sizeof(Nodes));
	hipMemcpy(simulation_snapshot, h_nodes.cr_state, sizeof(hiprandState_t) * n_nodes, hipMemcpyDeviceToHost);
}


void free_nodes() {
	Nodes h_nodes;
	hipMemcpyFromSymbol(&h_nodes, HIP_SYMBOL(nodes), sizeof(Nodes));
	hipFree(h_nodes.cr_state);
}

__device__
void set_model_params(int params[], uint n_params) {
	population = params[0];
	lookahead = params[1];
	mean = params[2];
}

__device__
int get_lookahead() {
	return lookahead;
}

__device__
void init_node(uint nid) {
	hiprand_init(nid, 0, 0, &(nodes.cr_state[nid]));

	uint n_events = population / g_n_nodes;
	if (nid < population % g_n_nodes) { n_events += 1; }

	for (uint i = 0; i < n_events; i++) {
		Event event;
		event.type = 1;
		event.sender = nid;
		event.receiver = nid;
		event.timestamp = i;

		append_event_to_queue(&event);
	}
}


__device__
void reinit_node(uint nid, int gvt) {
	
	uint n_events = population / g_n_nodes;
	hiprandState_t *cr_state = &(nodes.cr_state[nid]);

	for (uint i = 0; i < n_events; i++) {
		Event new_event;
		new_event.type = 1;
		new_event.sender = nid;
		new_event.receiver = random(cr_state, g_n_nodes);
		new_event.timestamp = gvt + lookahead + random_exp(cr_state, mean);

		char res = append_event_to_queue(&new_event);
	}
}

__device__ // private
char handle_event_type_1(Event *event) {
	uint nid = event->receiver;

#if OPTM_SYNC == 1
	uint lpid = nid / g_nodes_per_lp;

	if (state_queue_is_full(lpid)) { return 12; }
	if (antimsg_queue_is_full(lpid)) { return 13; }
#endif

	hiprandState_t *cr_state = &(nodes.cr_state[nid]);

	State old_state;
	old_state.cr_state = *cr_state;

	Event new_event;
	new_event.type = 1;
	new_event.sender = nid;
	new_event.receiver = random(cr_state, g_n_nodes);
	new_event.timestamp = event->timestamp + lookahead +
		random_exp(cr_state, mean);

	char res = append_event_to_queue(&new_event);

	if (res == 0) {
		nodes.cr_state[nid] = old_state.cr_state;
		return 11;
	}

#if OPTM_SYNC == 1
	append_state_to_queue(&old_state, lpid);
	append_antimsg_to_queue(&new_event);
#endif

	return 1;
}

#if OPTM_SYNC == 1
__device__ // private
void reverse_event_type_1(Event *event) {
	uint nid = event->receiver;
	uint lpid = nid / g_nodes_per_lp;

	State *old_state = delete_last_state(lpid);
	nodes.cr_state[nid] = old_state->cr_state;

	Event *antimsg = delete_last_antimsg(lpid);
	undo_event(antimsg);
}
#endif

__device__
char handle_event(Event *event) {
	uint type = event->type;

	if (type == 1) {
		return handle_event_type_1(event);
	} else {
		return 0;
	}
}

#if OPTM_SYNC == 1
__device__
void roll_back_event(Event *event) {
	uint type = event->type;

	if (type == 1) {
		reverse_event_type_1(event);
	}
}

__device__
uint get_number_states(Event *event) {
	return 1;
}

__device__
uint get_number_antimsgs(Event *event) {
	return 1;
}
#endif

__device__
void collect_statistics(uint nid) {
	return;
}

__device__
void print_statistics() {
	printf("STATISTICS NOT AVAILABLE\n");
}

extern "C" {
#include <core/core.h>
#include <lp/expose_lp_state.h>
extern void process_device_align_msg(unsigned lid, simtime_t time);
}

extern "C" void align_device_to_host_parallel(unsigned rid){
	int start = -1;
	int i;

	for(i=0;i<global_config.lps;i++){
		if(lid_to_rid(i) != rid) continue;
		if(lid_to_rid(i) != rid && start != -1) break;
		if(start == -1) start = i;
		hiprandState_t *state = (hiprandState_t*) get_lp_state_base_pointer(i);
		simulation_snapshot[i] = *state;
	}
	//printf("copying data from SIM to HOST by %u from %u to %u\n", rid, start, i);
}


extern "C" void align_device_to_host(int gvt, unsigned n_blocks, unsigned threads_per_block){
	copy_nodes_from_host(global_config.lps);
	hipDeviceSynchronize();
	printf("aligned memory from HOST to DEVICE\n");
	
	kernel_init_queues<<<n_blocks, threads_per_block>>>();
	hipDeviceSynchronize();
	printf("re init queues \n");
	
	kernel_reinit_nodes<<<n_blocks, threads_per_block>>>(gvt);
	hipDeviceSynchronize();
	printf("re init nodes \n");
	
	kernel_sort_event_queues<<<n_blocks, threads_per_block>>>();
	hipDeviceSynchronize();
	printf("sort queues \n");

}


extern "C" void align_host_to_device(int gvt){
	copy_nodes_to_host(global_config.lps);  
	hipDeviceSynchronize();
}



extern "C" void align_host_to_device_parallel(int gvt){
	int start = -1;
	int i;
	for(i=0;i<global_config.lps;i++){
		if(lid_to_rid(i) != rid && start == -1) continue;
		if(lid_to_rid(i) != rid && start != -1) break;
		if(start == -1) start = i;
		hiprandState_t *state = (hiprandState_t*) get_lp_state_base_pointer(i);
		*state = simulation_snapshot[i];
		//process_device_align_msg(i, gvt);
	}

	//printf("copying data from HOST to SIM by %u from %u to %u\n", rid, start, i);
}

