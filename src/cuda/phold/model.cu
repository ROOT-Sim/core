#include "hip/hip_runtime.h"
/*  This program is free software: you can redistribute it and/or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or
	(at your option) any later version.

	This program is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
	GNU General Public License for more details.

	You should have received a copy of the GNU General Public License
	along with this program.  If not, see <http://www.gnu.org/licenses/>. */


#include <cuda/cuda_gpu.h>
#include <cuda/queues.h>
#include <cuda/kernels.h>

#include <cuda/random.h>
#include "model.h"
#include "settings.h"


__device__ static Nodes nodes;
__device__ static uint	population;
__device__ static int	lookahead;
__device__ static int	mean;


extern uint events_per_node;
extern __device__ EQs	eq;
extern "C" uint get_n_nodes();
extern "C" uint get_n_lps();
extern "C" uint get_n_nodes_per_lp();
extern "C" uint get_n_blocks();



hiprandState_t *simulation_snapshot;
uint *sim_bo;
uint *sim_so;
uint *sim_uo;
uint *sim_ql;
Event *sim_events;

char malloc_nodes(uint n_nodes) {
	hipError_t err;

	Nodes h_nodes;
	simulation_snapshot = (hiprandState_t*) malloc(sizeof(hiprandState_t)*n_nodes);
	if(!sim_bo) sim_bo = (uint*)malloc(sizeof(uint) * n_nodes);
	if(!sim_so) sim_so = (uint*)malloc(sizeof(uint) * n_nodes);
	if(!sim_uo) sim_uo = (uint*)malloc(sizeof(uint) * n_nodes);
	if(!sim_ql) sim_ql = (uint*)malloc(sizeof(uint) * n_nodes);
	if(!sim_events) sim_events = (Event*)malloc(sizeof(Event) * n_nodes * events_per_node);

	if(!simulation_snapshot) {printf("no memory for HOST side model state\n"); exit(1); }

	err = hipMalloc(&(h_nodes.cr_state), sizeof(hiprandState_t) * n_nodes);
	if (err != hipSuccess) { return 0; }
	hipMemcpyToSymbol(HIP_SYMBOL(nodes), &h_nodes, sizeof(Nodes));

	return 1;
}

void free_nodes() {
	Nodes h_nodes;
	hipMemcpyFromSymbol(&h_nodes, HIP_SYMBOL(nodes), sizeof(Nodes));
	hipFree(h_nodes.cr_state);
}

__device__
void set_model_params(int params[], uint n_params) {
	population = params[0];
	lookahead = params[1];
	mean = params[2];
}

__device__
int get_lookahead() {
	return lookahead;
}

__device__
void init_node(uint nid) {
	hiprand_init(nid, 0, 0, &(nodes.cr_state[nid]));

	uint n_events = population / g_n_nodes;
	if (nid < population % g_n_nodes) { n_events += 1; }

	for (uint i = 0; i < n_events; i++) {
		Event event;
		event.type = 1;
		event.sender = nid;
		event.receiver = nid;
		event.timestamp = i;

		append_event_to_queue(&event);
	}
}


__device__
void reinit_node(uint nid, int gvt) {

	uint n_events = population / g_n_nodes;
	hiprandState_t *cr_state = &(nodes.cr_state[nid]);

	for (uint i = 0; i < n_events; i++) {
		Event new_event;
		new_event.type = 1;
		new_event.sender = nid;
		new_event.receiver = random(cr_state, g_n_nodes);
		new_event.timestamp = gvt + lookahead + random_exp(cr_state, mean);

		char res = append_event_to_queue(&new_event);
	}
}

__device__
static int current_gpu_model_phase = 0;

__device__
static uint get_receiver(uint me, hiprandState_t *cr_state, int now)
{
	int hot = (now / PHASE_WINDOW_SIZE) % 2;
    if(me == 0 && hot == 0 && current_gpu_model_phase == 1) {
	    current_gpu_model_phase = 0;
	    printf("GPU: ENTER HOT PHASE at wall clock time %f\n", 0.);
    } else if(me == 0 && hot == 1 && current_gpu_model_phase == 0) {
	    current_gpu_model_phase = 1;
	    printf("GPU: ENTER HOT PHASE at wall clock time %f\n", 0.);
    }

    if(current_gpu_model_phase == 0)
	    return random(cr_state, HOT_FRACTION * g_n_nodes);
    return random(cr_state, g_n_nodes);
}

__device__ // private
char handle_event_type_1(Event *event) {
	uint nid = event->receiver;

#if OPTM_SYNC == 1
	uint lpid = nid / g_nodes_per_lp;

	if (state_queue_is_full(lpid)) { return 12; }
	if (antimsg_queue_is_full(lpid)) { return 13; }
#endif

	hiprandState_t *cr_state = &(nodes.cr_state[nid]);

	State old_state;
	old_state.cr_state = *cr_state;

	Event new_event;
	new_event.type = 1;
	new_event.sender = nid;
	new_event.receiver = get_receiver(nid, cr_state, event->timestamp);
	new_event.timestamp = event->timestamp + lookahead +
		random_exp(cr_state, mean);

	char res = append_event_to_queue(&new_event);

	if (res == 0) {
		nodes.cr_state[nid] = old_state.cr_state;
		return 11;
	}

#if OPTM_SYNC == 1
	append_state_to_queue(&old_state, lpid);
	append_antimsg_to_queue(&new_event);
#endif

	return 1;
}

#if OPTM_SYNC == 1
__device__ // private
void reverse_event_type_1(Event *event) {
	uint nid = event->receiver;
	uint lpid = nid / g_nodes_per_lp;

	State *old_state = delete_last_state(lpid);
	nodes.cr_state[nid] = old_state->cr_state;

	Event *antimsg = delete_last_antimsg(lpid);
	undo_event(antimsg);
}
#endif

__device__
char handle_event(Event *event) {
	uint type = event->type;

	if (type == 1) {
		return handle_event_type_1(event);
	} else {
		return 0;
	}
}

#if OPTM_SYNC == 1
__device__
void roll_back_event(Event *event) {
	uint type = event->type;

	if (type == 1) {
		reverse_event_type_1(event);
	}
}

__device__
uint get_number_states(Event *event) {
	return 1;
}

__device__
uint get_number_antimsgs(Event *event) {
	return 1;
}
#endif

__device__
void collect_statistics(uint nid) {
	return;
}

__device__
void print_statistics() {
	printf("STATISTICS NOT AVAILABLE\n");
}

extern "C" {
#include <core/core.h>
#include <lp/expose_lp_state.h>
extern void process_device_align_msg(unsigned lid, simtime_t time);

}



void copy_nodes_from_host(uint n_nodes) {
	Nodes h_nodes;
	hipMemcpyFromSymbol(&h_nodes, HIP_SYMBOL(nodes), sizeof(Nodes));
	hipMemcpy(h_nodes.cr_state, simulation_snapshot, sizeof(hiprandState_t) * n_nodes, hipMemcpyHostToDevice);

	EQs h_eq;
	hipMemcpyFromSymbol(&h_eq, HIP_SYMBOL(eq), sizeof(EQs));
	hipMemcpy(h_eq.bo,sim_bo, sizeof(uint) * n_nodes, hipMemcpyHostToDevice);
	hipMemcpy(h_eq.so,sim_so, sizeof(uint) * n_nodes, hipMemcpyHostToDevice);
	hipMemcpy(h_eq.uo,sim_uo, sizeof(uint) * n_nodes, hipMemcpyHostToDevice);
	hipMemcpy(h_eq.ql,sim_ql, sizeof(uint) * n_nodes, hipMemcpyHostToDevice);
	hipMemcpy(h_eq.events,  sim_events, sizeof(Event) * n_nodes * events_per_node, hipMemcpyHostToDevice);

}


void copy_nodes_to_host(uint n_nodes) {
	Nodes h_nodes;
	hipMemcpyFromSymbol(&h_nodes, HIP_SYMBOL(nodes), sizeof(Nodes));
	hipMemcpy(simulation_snapshot, h_nodes.cr_state, sizeof(hiprandState_t) * n_nodes, hipMemcpyDeviceToHost);

	EQs h_eq;
	hipMemcpyFromSymbol(&h_eq, HIP_SYMBOL(eq), sizeof(EQs));

	hipMemcpy(sim_bo, h_eq.bo, sizeof(uint) * n_nodes, hipMemcpyDeviceToHost);
	hipMemcpy(sim_so, h_eq.so, sizeof(uint) * n_nodes, hipMemcpyDeviceToHost);
	hipMemcpy(sim_uo, h_eq.uo, sizeof(uint) * n_nodes, hipMemcpyDeviceToHost);
	hipMemcpy(sim_ql, h_eq.ql, sizeof(uint) * n_nodes, hipMemcpyDeviceToHost);
	hipMemcpy(sim_events, h_eq.events, sizeof(Event) * n_nodes * events_per_node, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	printf("transferring mem frm DEV t HST\n");

}



extern "C" int pack_and_insert_gpu_event(unsigned des_node, unsigned sen_node, int ts, unsigned type){
	uint gpu_lp = des_node/get_n_nodes_per_lp();
	uint idx = __sync_fetch_and_add(sim_ql + gpu_lp, 1);

	if(idx >= get_n_nodes_per_lp()*events_per_node){
		printf("adding more events that queue capacity\n");
		exit(1);
	}

	uint base = get_n_nodes_per_lp()*events_per_node*gpu_lp;

	Event *tgt = sim_events+base+idx;
	tgt->receiver = des_node;
	tgt->sender   = sen_node;
	tgt->timestamp = ts;
	tgt->type = type;
	return 1;
}


extern "C" void align_device_to_host_parallel_states(unsigned rid, simtime_t gvt){
	unsigned start;
	unsigned i;

	start = global_config.lps+1;
	for(i=0;i<global_config.lps;i++){
		if(lid_to_rid(i) != rid && start == (global_config.lps+1)) continue;
		if(lid_to_rid(i) != rid && start != (global_config.lps+1)) break;
		if(start == (global_config.lps+1)) start = i;
		align_lp_state_to_gvt(gvt,i);
		hiprandState_t *state = (hiprandState_t*) get_lp_state_base_pointer(i);
		simulation_snapshot[i] = *state;
	}

	clean_per_thread_queue();

	if(!rid){
		bzero(sim_events, sizeof(Event) * get_n_nodes() * events_per_node);
		bzero(sim_bo, sizeof(uint) * get_n_nodes());
		bzero(sim_so, sizeof(uint) * get_n_nodes());
		bzero(sim_uo, sizeof(uint) * get_n_nodes());
		bzero(sim_ql, sizeof(uint) * get_n_nodes());
	}

//	printf("A - copying events from SIM to HOST by %u from %u to %u \n", rid, start, i-1);
}




extern "C" void align_device_to_host_parallel_events(unsigned rid, simtime_t gvt){
	unsigned cnt_a = 0;
	unsigned cnt_c = 0;
	unsigned start = (global_config.lps+1);
	unsigned i;

	start = global_config.lps+1;
	cnt_a = 0;
	cnt_c = 0;
	for(i=0;i<global_config.lps;i++){
		if(lid_to_rid(i) != rid && start == (global_config.lps+1)) continue;
		if(lid_to_rid(i) != rid && start != (global_config.lps+1)) break;
		if(start == (global_config.lps+1)) start = i;

		cnt_a += estimate_transfer_per_lp_events_without_filter(i);
		cnt_c += transfer_per_lp_events(i,gvt);
	}

//	printf("B - copying events from SIM to HOST by %u from %u to %u : #events %u(%u)overall capacity %u\n", rid, start, i-1, cnt_c, cnt_a, events_per_node*get_n_nodes());

	transfer_per_thread_events(gvt);

}


extern "C" void align_device_to_host(unsigned threads_per_block){

	copy_nodes_from_host(global_config.lps);

	hipDeviceSynchronize();
	//printf("aligned memory from HOST to DEVICE\n");

	kernel_sort_event_queues<<<get_n_blocks(), threads_per_block>>>();
	hipDeviceSynchronize();
	//printf("sort queues \n");

}


extern "C" void align_host_to_device(){
	copy_nodes_to_host(global_config.lps);
	hipDeviceSynchronize();
}




extern "C" void align_host_to_device_parallel(simtime_t gvt){
	unsigned start = (global_config.lps+1);
	unsigned i;
	for(i=0;i<global_config.lps;i++){
		if(lid_to_rid(i) != rid && start == (global_config.lps+1)) continue;
		if(lid_to_rid(i) != rid && start != (global_config.lps+1)) break;
		if(start == (global_config.lps+1)) start = i;
		hiprandState_t *state = (hiprandState_t*) get_lp_state_base_pointer(i);
		*state = simulation_snapshot[i];
		process_device_align_msg(i, gvt);
	}
	//printf("copying states from HOST to SIM by %u from %u to %u\n", rid, start, i-1);

	uint pushed_events= 0;
	if(rid < get_n_lps()){
		for(i=0;i<get_n_lps()/global_config.n_threads;i++){
			uint lp = rid * (get_n_lps()/global_config.n_threads) + i;
			uint zero_idx  = lp*get_n_nodes_per_lp()*events_per_node;
			uint base_idx  = sim_bo[lp];
			uint start_idx = sim_so[lp];
			uint end_idx   = sim_uo[lp];
			//if(rid == 0) printf("base %u start %u end %u size %u\n", base_idx, start_idx, end_idx, get_n_nodes_per_lp()*events_per_node);
			while(start_idx != end_idx){
				uint effective = (base_idx+start_idx) % (get_n_nodes_per_lp()*events_per_node);
				Event *cur = sim_events+zero_idx+effective;
				//printf("A scheduling for %u a message from %u at %u\n", cur->receiver, cur->sender, cur->timestamp);
				custom_schedule_from_gpu(gvt, cur->sender, cur->receiver, (simtime_t) cur->timestamp, cur->type, NULL, 0);
				start_idx++;
				pushed_events++;
			}
		}
		//printf("copying events from HOST to SIM by %u from %u to %u GPU #LPS %u -- events pushed %u\n",
		//	rid, rid * (get_n_lps()/global_config.n_threads),rid * (get_n_lps()/global_config.n_threads)+get_n_lps()/global_config.n_threads-1, get_n_lps(), pushed_events);
	}

}
