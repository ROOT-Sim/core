/**
 * @file test/tests/integration/phold.c
 *
 * @brief A simple and stripped phold implementation
 *
 * SPDX-FileCopyrightText: 2008-2023 HPDCS Group <rootsim@googlegroups.com>
 * SPDX-License-Identifier: GPL-3.0-only
 */
extern "C" {

#include <ROOT-Sim.h>

#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include "cpu_curand.h"
}


#ifndef NUM_LPS
#define NUM_LPS 8192
#endif

#ifndef NUM_THREADS
#define NUM_THREADS 0
#endif

#define EVENT 1

unsigned int mean = 10000;

extern "C" {
static simtime_t lookahead = 0.0;

struct simulation_configuration conf;

void ProcessEvent(lp_id_t me, simtime_t now, unsigned event_type, const void *content, unsigned size,
    void *s)
{
	lp_id_t dest;
	hiprandState_t *state = (hiprandState_t *)s;
    unsigned int ts = 0;
    
	switch(event_type) {
		case LP_INIT:
			state = (hiprandState_t *)rs_malloc(sizeof(hiprandState_t));
			if(state == NULL) abort();
            cpu_curand_init(me, 0, 0, state);
			SetState(state);

            ts =  now + lookahead + cpu_random_exp(state, mean);
			ScheduleNewEvent(me, ts, EVENT, NULL, 0);
			break;

		case EVENT:
		case LP_REINIT:
			dest = cpu_random(state, conf.lps);
            ts =  now + lookahead + cpu_random_exp(state, mean);
			ScheduleNewEvent(dest, ts, EVENT, NULL, 0);
			break;

		case LP_FINI:
			break;
		default:
			fprintf(stderr, "Unknown event type\n");
			abort();
	}
}

bool CanEnd(lp_id_t me, const void *snapshot){ return false; }

}


int main(void)
{
    conf.lps = NUM_LPS,
    conf.n_threads = NUM_THREADS,
    conf.termination_time = 500000000,
    conf.gvt_period = 1000*250,
    conf.log_level = LOG_INFO,
    conf.stats_file = "phold",
    conf.ckpt_interval = 0,
    conf.core_binding = true,
    conf.serial = false,
    conf.use_gpu = true,
    conf.dispatcher = ProcessEvent,
    conf.committed = CanEnd,
	RootsimInit(&conf);
	return RootsimRun();
}

