#include "hip/hip_runtime.h"
/**
 * @file test/tests/integration/phold.c
 *
 * @brief A simple and stripped phold implementation
 *
 * SPDX-FileCopyrightText: 2008-2023 HPDCS Group <rootsim@googlegroups.com>
 * SPDX-License-Identifier: GPL-3.0-only
 */
extern "C" {

#include <ROOT-Sim.h>
#include <ftl/ftl.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include "cpu_curand.h"
#include "settings.h"
}


#ifndef NUM_LPS
#define NUM_LPS 8192
#endif

#ifndef NUM_THREADS
#define NUM_THREADS 0
#endif

#define EVENT 1

unsigned int mean = 10000;

extern "C" {
#include <unistd.h>
static simtime_t lookahead = 1000;

struct simulation_configuration conf;

static int hot_phase_count = 0;

static uint get_receiver(uint me, hiprandState_t *cr_state, int now)
{
	int cur_hot_phase = (now / PHASE_WINDOW_SIZE);
	int hot = cur_hot_phase%HOT_PHASE_PERIOD;
	
	if(me == 0){
		if(hot == 0 && cur_hot_phase > hot_phase_count){
			hot_phase_count = cur_hot_phase;
			printf("CPU: ENTER HOT PHASE at wall clock time %f %d\n", gimme_current_time_please(), hot);
		}
		else if(hot == 1 && cur_hot_phase > hot_phase_count){
			hot_phase_count = cur_hot_phase;
			printf("CPU: ENTER COLD PHASE at wall clock time %f %d\n", gimme_current_time_please(), hot);
		}
	}
	
	if(!(hot))
		return (unsigned int) cpu_random(cr_state, HOT_FRACTION * conf.lps)/(HOT_FRACTION);
	return (unsigned int) cpu_random(cr_state, conf.lps);
}


void ProcessEvent(lp_id_t me, simtime_t now, unsigned event_type, const void *content, unsigned size,
    void *s)
{
	lp_id_t dest;
	hiprandState_t *state = (hiprandState_t *)s;
    simtime_t ts = 0;
    int incr = 0;
	(void)content;
	(void)size;
	switch(event_type) {
		case LP_INIT:
			state = (hiprandState_t *)rs_malloc(sizeof(hiprandState_t));
			if(state == NULL) abort();
            cpu_curand_init(me, 0, 0, state);
			SetState(state);

			incr =  cpu_random_exp(state, mean);
            ts =  1.0*(now + lookahead + incr);
			ScheduleNewEvent(me, ts, EVENT, NULL, 0);
			break;

		case EVENT:
//			dest =  cpu_random(state, conf.lps);
			dest =  get_receiver(me, state, (int)now);
			incr =  cpu_random_exp(state, mean);
            ts =  1.0*(now + lookahead + incr);
			if(ts < now) printf("overflow ?? %d %f now %f\n", incr, ts, now);
			ScheduleNewEvent(dest, ts, EVENT, NULL, 0);
			break;

		case LP_REINIT:
		case LP_FINI:
			break;
		default:
			fprintf(stderr, "Unknown event type\n");
			abort();
	}
}

bool CanEnd(lp_id_t me, const void *snapshot){ (void)me; (void)snapshot; return false; }

}


int main(void)
{
    conf.lps = NUM_LPS,
    conf.n_threads = NUM_THREADS,
    conf.termination_time = END_SIM_GVT,
    conf.gvt_period = GVT_PERIOD,
    conf.log_level = LOG_INFO,
    conf.stats_file = "phold",
    conf.ckpt_interval = 0,
    conf.core_binding = true,
    conf.serial = false,
    conf.use_gpu = true,
    conf.dispatcher = ProcessEvent,
    conf.committed = CanEnd,
	RootsimInit(&conf);
	return RootsimRun();
}
