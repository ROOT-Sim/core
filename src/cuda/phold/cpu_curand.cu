#include "hip/hip_runtime.h"
extern "C" {

#include "cpu_curand.h"
    
#define CURAND_2POW32_INV (2.3283064e-10f)
    
    
void cpu_curand_init(unsigned long long seed, 
                                            unsigned long long subsequence, 
                                            unsigned long long offset, 
                                            hiprandStateXORWOW_t *state)
{
    // Break up seed, apply salt
    unsigned int s0 = ((unsigned int)seed) ^ 0xaad26b49UL;
    unsigned int s1 = (unsigned int)(seed >> 32) ^ 0xf7dcefddUL;
    // Simple multiplication to mix up bits
    unsigned int t0 = 1099087573UL * s0;
    unsigned int t1 = 2591861531UL * s1;
    state->d = 6615241 + t1 + t0;
    state->v[0] = 123456789UL + t0;
    state->v[1] = 362436069UL ^ t0;
    state->v[2] = 521288629UL + t1;
    state->v[3] = 88675123UL ^ t1;
    state->v[4] = 5783321UL + t0;
    state->boxmuller_flag = 0;
}

unsigned int cpu_curand(hiprandStateXORWOW_t *state){
    unsigned int t;
    t = (state->v[0] ^ (state->v[0] >> 2));
    state->v[0] = state->v[1];
    state->v[1] = state->v[2];
    state->v[2] = state->v[3];
    state->v[3] = state->v[4];
    state->v[4] = (state->v[4] ^ (state->v[4] <<4)) ^ (t ^ (t << 1));
    state->d += 362437;
    return state->v[4] + state->d;
}


unsigned int cpu_random(hiprandState_t *state, uint max) {
	return cpu_curand(state) % max;
}

float _cpu_curand_uniform(unsigned int x){
    return x * CURAND_2POW32_INV + (CURAND_2POW32_INV/2.0f);
}

float cpu_curand_uniform(hiprandStateXORWOW_t *state){
    return _cpu_curand_uniform(cpu_curand(state));
}


unsigned int cpu_random_exp(hiprandState_t *state, uint mean) {
	float ru = cpu_curand_uniform(state);
	return -(mean * logf(ru));
}


}
